#include "hip/hip_runtime.h"
#include <iostream>
#include <fstream>
#include <cmath>
#include <climits>

#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>

#include "common.hpp"
#include "ACO.cpp"
#include "TSP.cpp"
#include <float.h>

#include <hip/hip_cooperative_groups.h>
using namespace cooperative_groups;


#define DEBUG 0

__global__ 
void initCurand(hiprandStateXORWOW_t * state,
                const unsigned long seed,
                const uint32_t nAnts)
{
    const uint32_t idx = threadIdx.x + blockIdx.x * blockDim.x;
    if ( idx >= nAnts ) return;

    hiprand_init(seed, idx, 0, &state[idx]);
}

__device__ __forceinline__
float randXOR(hiprandState * state)
{
#if DEBUG 
    return 0.5f;
#else 
    return (float) hiprand_uniform(state);
#endif
}

__global__
void initialize(const float * distance,
                float * eta,
                float * pheromone,
                float * delta,
                const float initialPheromone,
                const uint32_t rows,
                const uint32_t cols)
{    
    const uint32_t row = blockIdx.y * blockDim.y + threadIdx.y;
    const uint32_t col = blockIdx.x * blockDim.x + threadIdx.x;
    if ( row >= rows || col >= cols ) return;

    const uint32_t idx = row * cols + col;
    const float d = distance[idx];
    if ( d == 0 ) {
        eta[idx] = 0.0f;
    } else {
        eta[idx] = 1.0f / d;
    }

    pheromone[idx] = initialPheromone;
    delta[idx] = 0.0f;
}

__global__
void calculateFitness(float * fitness,
                      const float * pheromone,
                      const float * eta,
                      const float alpha,
                      const float beta,
                      const uint32_t rows,
                      const uint32_t cols,
                      const uint32_t alignedCols)
{
    const uint32_t row = blockIdx.y * blockDim.y + threadIdx.y;
    const uint32_t col = blockIdx.x * blockDim.x + threadIdx.x;
    if ( row >= rows || col >= alignedCols ) return;

    const uint32_t idx = row * alignedCols + col;

    if (col < cols) {
        fitness[idx] = __powf(pheromone[row * cols + col], alpha) * __powf(eta[row * cols + col], beta);
    } else {
        fitness[idx] = 0.f;
    }
}

__device__ __forceinline__
float scanTileFloat(const thread_block_tile<32> & g, float x) {
    #pragma unroll
    for( uint32_t offset = 1 ; offset < 32 ; offset <<= 1 ) {
        float y = g.shfl_up(x, offset);
        if(g.thread_rank() >= offset) x += y;
    }
    return x;
}

__device__ __forceinline__
float maxTileFloat(const thread_block_tile<32> & g, float x) {
    
    #pragma unroll
    for ( uint32_t offset = 16; offset > 0; offset >>= 1 ) {
        const float y = g.shfl_xor(x, offset);
        x = fmaxf(x, y);
    }
    return x;
}



// __global__
// void claculateTour(uint32_t * tabu,
//                    const float * fitness,
//                    const uint32_t rows,
//                    const uint32_t cols,
//                    hiprandStateXORWOW_t * state)
// {
//     const uint32_t numberOfBlocks = (cols + 31) / 32;

//     extern __shared__ uint32_t smem[];
//     float    * p       = (float *)    smem;
//     uint32_t * k       = (uint32_t *) &p[cols];
//     float    * reduce  = (float *)    &k[1];
//     uint8_t * visited  = (uint8_t *)  &reduce[numberOfBlocks];

//     thread_block_tile<32> tile32 = tiled_partition<32>(this_thread_block());
//     const uint32_t tid = threadIdx.x;

//     // initialize visited array
//     for (uint32_t i = tid; i < cols; i += blockDim.x) { 
//         visited[i] = 1;
//     }
//     __syncthreads();

//     // get random starting city and update visited and tabu
//     if (tid == 0) {
//         const uint32_t kappa = cols * randXOR(state + blockIdx.x);
//         *k = kappa;
//         visited[kappa] = 0;
//         tabu[blockIdx.x * cols] = kappa;
//     }
//     __syncthreads();

//     for (uint32_t s = 1; s < cols; ++s) {
//         // get city from shared memory
//         const uint32_t kappa = *k;

//         // update probability values
//         for (uint32_t i = tid; i < cols; i += blockDim.x) { 
//             p[i] = fitness[kappa * cols + i] * visited[i];
//         }
//         __syncthreads();

//         for (uint32_t blockId = tid / 32; blockId < numberOfBlocks; blockId += blockDim.x / 32) {
//             const uint32_t warpTid = tile32.thread_rank() + (blockId * 32);
            
//             const float x = (warpTid < cols) ? p[warpTid] : 0.f;
//             const float y = scanTileFloat(tile32, x);
//             const float z = tile32.shfl(y, 31);
//             if (warpTid < cols) p[warpTid] = y / z;
//             if (tile32.thread_rank() == 0) reduce[blockId] = z;
//         }

//         __syncthreads();

//         if (tid < 32) {

//             uint32_t selectedBlock = 1234567890; // fake number just to be sure that will not appear somewere
//             float selectedMax = -1.f;

//             //TODO: verify if selectedBlock is correct when cols >= 1024
//             for (uint32_t stride = 0; stride < numberOfBlocks; stride += 32) {
//                 const uint32_t warpTid = tid + stride;
//                 const float x = (warpTid < numberOfBlocks) ? reduce[warpTid] : 0.f;
//                 selectedMax = fmaxf(x, selectedMax);
//                 selectedBlock = (x == selectedMax) ? warpTid : selectedBlock;

//                 const float y = maxTileFloat(tile32, selectedMax);
//                 const uint32_t mask = tile32.ballot( x == y );
//                 const uint32_t maxTile = __ffs(mask) - 1;
//                 selectedMax = tile32.shfl(y, maxTile);
//                 selectedBlock = tile32.shfl(selectedBlock, maxTile);
//             }

//             // generate and broadcast randomFloat
//             float randomFloat = -1.f;
//             if (tid == 0) {
//                 randomFloat = randXOR(state + blockIdx.x);
//             }
//             randomFloat = tile32.shfl(randomFloat, 0);
            
//             const uint32_t probabilityId = selectedBlock * 32 + tid;
//             if (probabilityId < cols) {
//                 const uint32_t bitmask = tile32.ballot(randomFloat < p[probabilityId]); 
//                 const uint32_t selected = __ffs(bitmask) - 1;

//                 if (tid == selected) {
//                     const uint32_t nextCity = selectedBlock * 32 + selected;
//                     tabu[blockIdx.x * cols + s] = nextCity;
//                     visited[nextCity] = 0;
//                     *k = nextCity;
//                 }
//             }
//         }
//         __syncthreads();
//     }
// }

#define FULL_MASK 0xFFFFFFFF
__device__ __forceinline__
float scanWarpFloat(const uint32_t tid, float x) {
    #pragma unroll
    for( uint32_t offset = 1 ; offset < 32 ; offset <<= 1 ) {
        const float y = __shfl_up_sync(FULL_MASK, x, offset);
        if(tid >= offset) x += y;
    }
    return x;
}

__global__
void claculateTour(uint32_t * tabu,
                   const float * fitness,
                   const uint32_t rows,
                   const uint32_t cols,
                   const uint32_t alignedCols,
                   hiprandStateXORWOW_t * state)
{
    extern __shared__ uint32_t smem[];
    float    * p = (float *)     smem;
    uint32_t * k = (uint32_t *) &p[alignedCols];
    uint8_t  * v = (uint8_t *)  &k[1]; 

    const uint32_t tid = threadIdx.x;
    const uint32_t bid = blockIdx.x;

    // initialize visited array
    for (uint32_t i = tid; i < alignedCols; i += 32) { 
        v[i] = 1;
    }
    __syncwarp();

    // get random starting city and update visited and tabu
    if (tid == 0) {
        const uint32_t kappa = cols * randXOR(state + bid);
        *k = kappa;
        v[kappa] = 0;
        tabu[blockIdx.x * cols] = kappa;
    }
    

    for (uint32_t s = 1; s < cols; ++s) {
        __syncwarp(); // sync warp once for tabu initialization and then for *k value update
        // get city from shared memory
        const uint32_t kappa = *k;

        // update probability values
        for (uint32_t pid = tid; pid < alignedCols; pid += 32) { 
            p[pid] = fitness[kappa * alignedCols + pid] * v[pid];
        }
        __syncwarp();

        float sum = 0.f;
        for (uint32_t pid = tid; pid < alignedCols; pid += 32) {
            const float x = p[pid];
            const float y = sum + scanWarpFloat(tid, x);
            p[pid] = y;
            sum = __shfl_sync(FULL_MASK, y, 31);
            // printf("%d) %f - %f - %f\n", pid, x, y, sum);
        }

        __syncwarp();

        // generate and broadcast randomFloat
        float randomFloat = -1.f;
        if (tid == 0) {
            randomFloat = randXOR(state + blockIdx.x);
        }
        randomFloat = __shfl_sync(FULL_MASK, randomFloat, 0);

        const float probability = randomFloat * sum;
        for (uint32_t pid = tid; pid < alignedCols; pid += 32) {
            
            const float prevP = (pid == 0 ? 0.f : p[pid - 1]);
            const float currP = p[pid];
            const float magicProbability = (prevP - probability) * (currP - probability);
            const uint32_t ballotMask = __ballot_sync(FULL_MASK, magicProbability <= 0.f);
            const uint32_t winner = __ffs(ballotMask);

            if (winner > 0) {
                if (tid == winner - 1) {
                    tabu[blockIdx.x * cols + s] = pid;
                    v[pid]= 0;
                    *k = pid;
                }
                break;
            }
        }
    }
}

__device__ __forceinline__
float reduceTileFloat(const thread_block_tile<32> & g, float x) {
    
    #pragma unroll
    for ( uint32_t offset = 16; offset > 0; offset >>= 1 ) {
        x += g.shfl_down(x, offset);
    }
    return x;
}

__global__
void calculateTourLen(const float * distance,
                      const uint32_t * tabu,
                      float * tourLen,
                      const uint32_t rows,
                      const uint32_t cols)
{
    __shared__ float finalLength[1];

    thread_block_tile<32> tile32 = tiled_partition<32>(this_thread_block());
    const uint32_t numberOfBlocks = (cols + 31) / 32;

    float totalLength = 0.f;
    for (uint32_t blockId = threadIdx.x / 32; blockId < numberOfBlocks; blockId += blockDim.x / 32) {
        const uint32_t warpTid = blockIdx.x * cols + tile32.thread_rank() + (blockId * 32);

        float len = 0.f;
        if ( tile32.thread_rank() + (blockId * 32) < cols - 1 ) {
            const uint32_t from = tabu[warpTid];
            const uint32_t to   = tabu[warpTid + 1];
            len  = distance[from * cols + to];
        }
        totalLength += reduceTileFloat(tile32, len);
    }

    if (threadIdx.x == 0) {
        const uint32_t from = tabu[blockIdx.x * cols + cols - 1];
        const uint32_t to   = tabu[blockIdx.x * cols];
        const float    len  = distance[from * cols + to];
        
        totalLength += len;

        finalLength[0] = 0.f;
    }

    __syncthreads();

    if (tile32.thread_rank() == 0) {
        atomicAdd(finalLength, totalLength);
    }

    __syncthreads();

    if (threadIdx.x == 0) {
        tourLen[blockIdx.x] = finalLength[0];
    }
}

__device__ __forceinline__
float minTileFloat(const thread_block_tile<32> & g, float x) {
    
    #pragma unroll
    for ( uint32_t offset = 16; offset > 0; offset >>= 1 ) {
        const float y = g.shfl_xor(x, offset);
        x = fminf(x, y);
    }
    return x;
}

__global__
void updateBest(uint32_t * bestPath,
                const uint32_t * tabu,
                const float * tourLen,
                const uint32_t rows,
                const uint32_t cols,
                float * bestPathLen)
{
    thread_block_tile<32> tile32 = tiled_partition<32>(this_thread_block());
    const uint32_t tid = threadIdx.x;

    uint32_t bestAnt = 1234567890; // fake number just to be sure that will not appear somewere
    float minLength = FLT_MAX;

    //TODO: verify if bestAnt is correct when cols >= 1024
    for (uint32_t stride = 0; stride < cols; stride += 32) {
        const uint32_t warpTid = tid + stride;
        const float x = (warpTid < cols) ? tourLen[warpTid] : FLT_MAX;
        minLength = fminf(x, minLength);
        bestAnt = (x == minLength) ? warpTid : bestAnt;

        const float y = minTileFloat(tile32, minLength);
        const uint32_t mask = tile32.ballot( x == y );
        const uint32_t maxTile = __ffs(mask) - 1;
        minLength = tile32.shfl(y, maxTile);
        bestAnt = tile32.shfl(bestAnt, maxTile);
    }

    for (uint32_t i = tid; i < cols; i += 32) {
        bestPath[i] = tabu[bestAnt * cols + i];
    }

    if (tid == 0) {
        bestPathLen[0] = minLength;
    }
}


__global__
void updateDelta(float * delta,
                 const uint32_t * tabu,
                 const float * tourLen,
                 const uint32_t rows,
                 const uint32_t cols,
                 const float q)
{
    extern __shared__ uint32_t tabus[];
    const uint32_t tid = threadIdx.x;

    for (uint32_t i = tid; i < cols; i += blockDim.x) { 
        tabus[i] = tabu[blockIdx.x * cols + i];
    }
    __syncthreads();

    const float tau = q / tourLen[blockIdx.x];

    for (uint32_t i = tid; i < cols - 1; i += blockDim.x) { 
        const uint32_t from = tabus[i];
        const uint32_t to   = tabus[i + 1];
        atomicAdd(delta + (from * cols + to), tau);
    }

    if (tid == 0) {
        const uint32_t from = tabus[cols - 1];
        const uint32_t to   = tabus[0];
        atomicAdd(delta + (from * cols + to), tau);
    }
}

__global__
void updatePheromone(float * pheromone,
                     float * delta,
                     const uint32_t rows,
                     const uint32_t cols,
                     const float rho)
{
    const uint32_t row = blockIdx.y * blockDim.y + threadIdx.y;
    const uint32_t col = blockIdx.x * blockDim.x + threadIdx.x;
    if ( row >= rows || col >= cols ) return;

    const uint32_t idx = row * cols + col;

    const float p = pheromone[idx];
    pheromone[idx] = p * (1.0f - rho) + delta[idx];
    delta[idx] = 0.0f;
}


uint32_t numberOfBlocks(uint32_t numberOfElements, uint32_t blockSize) {
    return (numberOfElements + blockSize - 1) / blockSize;
}

uint32_t roundWithBlockSize(uint32_t numberOfElements, uint32_t blockSize)
{
    return numberOfBlocks(numberOfElements, blockSize) * blockSize; 
}

#ifndef D_TYPE
#define D_TYPE float
#endif


int main(int argc, char * argv[]) {

	char * path = new char[MAX_LEN];
	D_TYPE alpha = 4.0f;
	D_TYPE beta = 2.0f;
	D_TYPE q = 55.0f;
	D_TYPE rho = 0.8f;
	int maxEpoch = 10;
	
	if (argc < 7) {
		cout << "Usage: ./acogpu file.tsp alpha beta q rho maxEpoch" << endl;
		exit(-1);
	}

	argc--;
	argv++;
    path     = argv[0];
    alpha    = parseArg<float>   (argv[1]);
    beta     = parseArg<float>   (argv[2]);
    q        = parseArg<float>   (argv[3]);
    rho      = parseArg<float>   (argv[4]);
    maxEpoch = parseArg<uint32_t>(argv[5]);

	TSP<D_TYPE> * tsp = new TSP<D_TYPE>(path);
	ACO<D_TYPE> * aco = new ACO<D_TYPE>(tsp->dimension, tsp->dimension, alpha, beta, q, rho, maxEpoch, 0);

#if DEBUG
    const uint32_t seed         = 123;
#else 
    const uint32_t seed         = time(0);
#endif
    const uint32_t nAnts        = aco->nAnts;
    const uint32_t nCities      = tsp->dimension;
    const float    valPheromone = 1.0f / nCities;

    hiprandStateXORWOW_t * state;
    float * distance;
    float * eta;
    float * pheromone;
    float * fitness;
    float * delta;
    uint32_t * tabu;
    float * tourLen;
    uint32_t * bestPath;
    float * bestPathLen;

    const uint32_t alignedAnts = ((nAnts + 31) / 32) * 32;
    const uint32_t alignedCities = ((nCities + 31) / 32) * 32;
    const uint32_t alignedFitnessElems = nCities * alignedCities;

    std::cout << "alignedCities " << alignedCities << std::endl;

    const uint32_t elems = nCities * nCities;
    hipMallocManaged(&state,       nAnts * sizeof(hiprandStateXORWOW_t));
    hipMallocManaged(&distance,    elems * sizeof(float));
    hipMallocManaged(&eta,         elems * sizeof(float));
    hipMallocManaged(&pheromone,   elems * sizeof(float));
    hipMallocManaged(&fitness,     alignedFitnessElems * sizeof(float));
    hipMallocManaged(&delta,       elems * sizeof(float));
    hipMallocManaged(&tabu,        nAnts * nCities * sizeof(uint32_t));
    hipMallocManaged(&tourLen,     nAnts * sizeof(float));
    hipMallocManaged(&bestPath,    nCities * sizeof(uint32_t));
    hipMallocManaged(&bestPathLen, sizeof(float));

    const uint32_t totalMemory = (2 * nAnts + elems * 4 + alignedFitnessElems + nAnts * nCities + nCities) * sizeof(uint32_t);
    std::cout << " *** totalMemory **** \t" << (totalMemory / 1024.f) / 1024.f << "MB" << std::endl;


    *bestPathLen = INT_MAX;

    for (uint32_t i = 0; i < nCities; ++i) {
        for (uint32_t j = 0; j < nCities; ++j) {
            distance[i * nCities + j] = tsp->edges[i * nCities +j];
        }
    }

    dim3 dimBlock1D(32);
    dim3 dimBlock2D(16, 16);

    dim3 gridAnt1D(numberOfBlocks(nAnts, dimBlock1D.x));
    dim3 gridMatrix2D(numberOfBlocks(nCities, dimBlock2D.y), numberOfBlocks(nCities, dimBlock2D.x));

    startTimer();

    initCurand<<<gridAnt1D,    dimBlock1D>>>(state, seed, nAnts);
    initialize<<<gridMatrix2D, dimBlock2D>>>(distance, eta, pheromone, delta, valPheromone, nCities, nCities);

    const dim3 fitGrid(alignedCities / 16, alignedCities / 16);
    const dim3 fitBlock(16, 16);

    const dim3 tourGrid(nAnts); // number of blocks
    const dim3 tourBlock(32); // number of threads in a block
    const uint32_t alignedCols = ((nCities + 31) / 32) * 32;
    const uint32_t tourShared  = alignedCols  * sizeof(float)    + // p
                                 1            * sizeof(uint32_t) + // k
                                 alignedCols  * sizeof(uint8_t);   // v
    std::cout << " *** TOUR  sharedMemory **** \tKB " << (tourShared / 1024.f) << std::endl;

    const dim3 lenGrid(nAnts);
    const dim3 lenBlock(64);
    const uint32_t lenShared = lenBlock.x / 32 * sizeof(float);
    std::cout << " *** LEN   sharedMemory **** \tKB " << (lenShared / 1024.f)  << std::endl;

    const dim3 bestGrid(1);
    const dim3 bestBlock(32);

    const dim3 deltaGrid(nAnts);
    const dim3 deltaBlock(32);
    const uint32_t deltaShared = nCities * sizeof(uint32_t);
    std::cout << " *** DELTA sharedMemory **** \tKB " << (deltaShared / 1024.f) << std::endl;

    uint32_t epoch = 0;
    do {
        calculateFitness <<<fitGrid,      fitBlock               >>>(fitness, pheromone, eta, alpha, beta, nCities, nCities, alignedCities);
        claculateTour    <<<tourGrid,     tourBlock,  tourShared >>>(tabu, fitness, nAnts, nCities, alignedCols, state);
        calculateTourLen <<<lenGrid,      lenBlock,   lenShared  >>>(distance, tabu, tourLen, nAnts, nCities);
        updateBest       <<<bestGrid,     bestBlock              >>>(bestPath, tabu, tourLen, nAnts, nCities, bestPathLen);
        updateDelta      <<<deltaGrid,    deltaBlock, deltaShared>>>(delta, tabu, tourLen, nAnts, nCities, q);
        updatePheromone  <<<gridMatrix2D, dimBlock2D             >>>(pheromone, delta, nCities, nCities, rho);
    } while (++epoch < maxEpoch);

    hipDeviceSynchronize();
    
    // printMatrix("fitness", fitness, nCities, alignedCities);
    // printMatrix("tabu", tabu, nAnts, nCities);

    stopAndPrintTimer();
    cout << (tsp->checkPath(bestPath) == 1 ? "Path OK!" : "Error in the path!") << endl;
    cout << "bestPathLen: " << *bestPathLen << endl;
    cout << "CPU Path distance: " << tsp->calculatePathLen(bestPath) << endl;
    printMatrix("bestPath", bestPath, 1, nCities);


    hipFree(state);
    hipFree(distance);
    hipFree(eta);
    hipFree(pheromone);
    hipFree(fitness);
    hipFree(delta);
    hipFree(tabu);
    hipFree(tourLen);
    hipFree(bestPath);
    hipFree(bestPathLen);

    return 0;
}
