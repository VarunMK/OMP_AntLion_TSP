#include "hip/hip_runtime.h"
#include <iostream>
#include <fstream>
#include <float.h>
#include <cmath>
#include <climits>
#include <thread>
#include <chrono>

#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>
#include <hip/hip_cooperative_groups.h>
using namespace cooperative_groups;

#include "common.hpp"
#include "TSP.cpp"

#ifndef D_TYPE
#define D_TYPE float
#endif

#define cudaCheck(ans) do { cudaAssert((ans), __FILE__, __LINE__); } while(0)
inline void cudaAssert(hipError_t code, const char * file, uint32_t line, bool abort = true)
{
    if (code != hipSuccess) {
        std::clog <<  "hipErrorAssert: "<< hipGetErrorString(code) << " " << file << " " << line << std::endl;
        if (abort) { 
            exit(code);
        }
    }
}


__global__ 
void initCurand(hiprandStateXORWOW_t * state,
                const uint64_t seed,
                const uint32_t elems)
{
    const uint32_t tid = blockIdx.x * blockDim.x + threadIdx.x;

    for (uint32_t c = tid; c < elems; c += gridDim.x * blockDim.x) {
        hiprand_init(seed, c, 0, &state[c]);
    }
}

__device__ __forceinline__
float randXOR(hiprandState * state)
{
    return (float) hiprand_uniform(state);
}

__global__
void initEta(float * eta,
             const float * edges,
             const uint32_t rows,
             const uint32_t cols)
{
    const uint32_t tid = blockIdx.x * blockDim.x + threadIdx.x;

    for (uint32_t r = 0; r < rows; ++r) {
        for (uint32_t c = tid; c < cols; c += gridDim.x * blockDim.x) {
            const uint32_t id = r * cols + c;
            const float d = edges[id];
            eta[id] = (d == 0.0) ? 0.0 : 1.0 / d;
        }
    }
}

__global__
void initPheromone(float * pheromone,
                   const float initialValue,
                   const uint32_t rows,
                   const uint32_t cols,
                   const uint32_t realCols)
{
    const uint32_t tid = blockIdx.x * blockDim.x + threadIdx.x;

    for (uint32_t r = 0; r < rows; ++r) {
        for (uint32_t c = tid; c < cols; c += gridDim.x * blockDim.x) {
            const uint32_t id = r * cols + c;
            pheromone[id] = initialValue * ( c < realCols );
        }
    }
}

__global__
void initDelta(float * delta,
               const uint32_t rows,
               const uint32_t cols)
{
    const uint32_t tid = blockIdx.x * blockDim.x + threadIdx.x;

    for (uint32_t r = 0; r < rows; ++r) {
        for (uint32_t c = tid; c < cols; c += gridDim.x * blockDim.x) {
        const uint32_t id = r * cols + c;
            delta[id] = 0.0;
        }
    }
}

__global__
void calcFitness(float * fitness,
                 const float * pheromone,
                 const float * eta,
                 const float alpha,
                 const float beta,
                 const uint32_t rows,
                 const uint32_t cols)
{
    const uint32_t tid = blockIdx.x * blockDim.x + threadIdx.x;

    for (uint32_t r = 0; r < rows; ++r) {
        for (uint32_t c = tid; c < cols; c += gridDim.x * blockDim.x) {
            const uint32_t id = r * cols + c;
            const float p = pheromone[id];
            const float e = eta[id];
            fitness[id] = __powf(p, alpha) * __powf(e, beta);
        }
    }
}

#define FULL_MASK 0xFFFFFFFF
__device__ __forceinline__
float scanWarpFloat(const uint32_t tid, float x) {
    #pragma unroll
    for( uint32_t offset = 1 ; offset < 32 ; offset <<= 1 ) {
        const float y = __shfl_up_sync(FULL_MASK, x, offset);
        if(tid >= offset) x += y;
    }
    return x;
}

__global__
void calcTour(uint32_t * tabu,
              const float * fitness,
              const uint32_t rows,
              const uint32_t cols,
              const uint32_t alignedCols,
              hiprandStateXORWOW_t * state)
{
    extern __shared__ uint32_t smem[];
    float    * p = (float *)     smem;
    uint32_t * k = (uint32_t *) &p[alignedCols];
    uint8_t  * v = (uint8_t *)  &k[1]; 

    const uint32_t tid = threadIdx.x;

    for (uint32_t ant = blockIdx.x; ant < rows; ant += gridDim.x) {

        for (uint32_t i = tid; i < alignedCols; i += 32) {
            v[i] = 1;
        }
        __syncwarp();

        if (tid == 0) {
            const uint32_t kappa = cols * randXOR(state + ant);
            *k = kappa;
            v[kappa] = 0;
            tabu[ant * alignedCols] = kappa;
        }
        

        for (uint32_t s = 1; s < cols; ++s) {
            __syncwarp(); // sync warp once for tabu initialization and then for *k value update
            // get city from shared memory
            const uint32_t kappa = *k;

            for (uint32_t pid = tid; pid < alignedCols; pid += 32) {
                p[pid] =  fitness[kappa * alignedCols + pid] * v[pid];
            }
            __syncwarp();

            float sum = 0.0;
            for (uint32_t pid = tid; pid < alignedCols; pid += 32) {
                const float x = p[pid];
                const float y = sum + scanWarpFloat(tid, x);
                p[pid] = y;
                sum = __shfl_sync(FULL_MASK, y, 31);
            }
            __syncwarp();

            float randomFloat = -1.0;
            if (tid == 0) {
                randomFloat = randXOR(state + ant);
            }
            randomFloat = __shfl_sync(FULL_MASK, randomFloat, 0);

            const float probability = randomFloat * sum;
            for (uint32_t pid = tid; pid < alignedCols; pid += 32) {
                
                // const float prevP = (pid == 0 ? 0.0 : p[pid - 1]);
                const float currP = p[pid];
                // const float magicProbability = (prevP - probability) * (currP - probability);
                // const float magicProbability = currP - probability;
                const uint32_t ballotMask = __ballot_sync(FULL_MASK,  probability <= currP);
                const uint32_t winner = __ffs(ballotMask);

                if (winner > 0) {
                    if (tid == winner - 1) {
                        tabu[ant * alignedCols + s] = pid;
                        v[pid]= 0;
                        *k = pid;
                    }
                    break;
                }
            }
        }
    }
}

__device__ __forceinline__
float reduceTileFloat(const thread_block_tile<32> & g, float x) {
    
    #pragma unroll
    for ( uint32_t offset = 16; offset > 0; offset >>= 1 ) {
        x += g.shfl_down(x, offset);
    }
    return x;
}

__global__
void calcTourLength(float * tourLength,
                    const float * edges,
                    const uint32_t * tabu,
                    const uint32_t rows,
                    const uint32_t cols,
                    const uint32_t realCols)
{
    __shared__ float finalLength[1];

    thread_block_tile<32> tile32 = tiled_partition<32>(this_thread_block());
    const uint32_t numberOfBlocks = (cols + 31) / 32;

    float totalLength = 0.0;
    for (uint32_t blockId = threadIdx.x / 32; blockId < numberOfBlocks; blockId += blockDim.x / 32) {
        const uint32_t warpTid = blockIdx.x * cols + tile32.thread_rank() + (blockId * 32);

        float len = 0.0;
        if (tile32.thread_rank() + (blockId * 32) < realCols - 1) {
            const uint32_t from = tabu[warpTid];
            const uint32_t to   = tabu[warpTid + 1];
            len  = edges[from * cols + to];
        }
        totalLength += reduceTileFloat(tile32, len);
    }

    if (threadIdx.x == 0) {
        const uint32_t from = tabu[blockIdx.x * cols + realCols - 1];
        const uint32_t to   = tabu[blockIdx.x * cols];
        const float    len  = edges[from * cols + to];
        
        totalLength += len;

        finalLength[0] = 0.0;
    }
    __syncthreads();

    if (tile32.thread_rank() == 0) {
        atomicAdd(finalLength, totalLength);
    }
    __syncthreads();

    if (threadIdx.x == 0) {
        tourLength[blockIdx.x] = finalLength[0];
    }
}

__device__ __forceinline__
float minTileFloat(const thread_block_tile<32> & g, float x) {
    
    #pragma unroll
    for ( uint32_t offset = 16; offset > 0; offset >>= 1 ) {
        const float y = g.shfl_xor(x, offset);
        x = fminf(x, y);
    }
    return x;
}

__global__
void updateBestTour(uint32_t * bestTour,
                    float * bestTourLength,
                    const uint32_t * tabu,
                    const float * tourLength,
                    const uint32_t rows,
                    const uint32_t cols,
                    const uint32_t realCols)
{
    thread_block_tile<32> tile32 = tiled_partition<32>(this_thread_block());
    const uint32_t tid = threadIdx.x;

    uint32_t bestAnt = 1234567890; // fake number just to be sure that will not appear somewhere
    float minLength = FLT_MAX;

    for (uint32_t stride = 0; stride < cols; stride += 32) {
        const uint32_t warpTid = tid + stride;
        const float x = (warpTid < realCols) ? tourLength[warpTid] : FLT_MAX; //TODO: find a way to avoid realCols parameter
        minLength = fminf(x, minLength);
        bestAnt = (x == minLength) ? warpTid : bestAnt;

        const float y = minTileFloat(tile32, minLength);
        const uint32_t mask = tile32.ballot( x == y );
        const uint32_t maxTile = __ffs(mask) - 1;
        minLength = tile32.shfl(y, maxTile);
        bestAnt = tile32.shfl(bestAnt, maxTile);
    }

    for (uint32_t i = tid; i < cols; i += 32) {
        bestTour[i] = tabu[bestAnt * cols + i];
    }

    if (tid == 0) {
        bestTourLength[0] = minLength;
    }
}

__global__
void updateDelta(float * delta,
                 const uint32_t * tabu,
                 const float * tourLenght,
                 const uint32_t rows,
                 const uint32_t cols,
                 const uint32_t realCols,
                 const float q)
{
    extern __shared__ uint32_t tabus[];
    const uint32_t tid = threadIdx.x;

    for (uint32_t i = tid; i < cols; i += blockDim.x) {
        tabus[i] = tabu[blockIdx.x * cols + i];
    }
    __syncthreads();

    const float tau = q / tourLenght[blockIdx.x];

    for (uint32_t i = tid; i < realCols - 1; i += blockDim.x) { 
        const uint32_t from = tabus[i];
        const uint32_t to   = tabus[i + 1];
        atomicAdd(delta + (from * cols + to), tau);
        atomicAdd(delta + (to * cols + from), tau);
    }

    if (tid == 0) {
        const uint32_t from = tabus[realCols - 1];
        const uint32_t to   = tabus[0];
        atomicAdd(delta + (from * cols + to), tau);
        atomicAdd(delta + (to * cols + from), tau);
    }
}

__global__
void updatePheromone(float * pheromone,
                     const float * delta,
                     const uint32_t rows,
                     const uint32_t cols,
                     const float rho)
{
    const uint32_t tid = blockIdx.x * blockDim.x + threadIdx.x;

    for (uint32_t r = 0; r < rows; ++r) {
        for (uint32_t c = tid; c < cols; c += gridDim.x * blockDim.x) {
            const uint32_t id = r * cols + c;
            const float p = pheromone[id];
            pheromone[id] = p * (1.0 - rho) + delta[id];
        }
    }
}

inline uint32_t numberOfBlocks(const uint32_t elems, const uint32_t blockSize) {
    return (elems + blockSize - 1) / blockSize;
}

inline uint32_t alignToWarp(const uint32_t elems) {
    return ((elems + 31) / 32) * 32;
}


int main(int argc, char * argv[]) {

	char * path = new char[MAX_LEN];
	D_TYPE alpha = 1.0f;
	D_TYPE beta = 2.0f;
	D_TYPE q = 1.0f;
	D_TYPE rho = 0.5f;
	int maxEpoch = 1;
	
	if (argc < 7) {
		cout << "Usage: ./acogpu file.tsp alpha beta q rho maxEpoch" << endl;
		exit(-1);
	}

	argc--;
	argv++;
    path     = argv[0];
    alpha    = parseArg<float>   (argv[1]);
    beta     = parseArg<float>   (argv[2]);
    q        = parseArg<float>   (argv[3]);
    rho      = parseArg<float>   (argv[4]);
    maxEpoch = parseArg<uint32_t>(argv[5]);

    TSP<D_TYPE> tsp(path);

#if DEBUG
    const uint64_t seed         = 123;
#else 
    const uint64_t seed         = time(0);
#endif
    const uint32_t nAnts        = tsp.getNCities();
    const uint32_t nCities      = tsp.getNCities();
    const float    valPheromone = 1.0f / nCities;

    hiprandStateXORWOW_t * randState;
    float * edges;
    float * eta;
    float * pheromone;
    float * fitness;
    float * delta;
    uint32_t * tabu;
    float * tourLength;
    uint32_t * bestTour;
    float * bestTourLength;

    const uint32_t alignedAnts = alignToWarp(nAnts);
    const uint32_t alignedCities = alignToWarp(nCities);

    const uint32_t randStateElems  = alignedAnts;
    const uint32_t edgesElems      = nCities * alignedCities;
    const uint32_t etaElems        = nCities * alignedCities;
    const uint32_t pheromoneElems  = nCities * alignedCities;
    const uint32_t fitnessElems    = nCities * alignedCities;
    const uint32_t deltaElems      = nCities * alignedCities;
    const uint32_t tabuElems       = nAnts   * alignedCities;
    const uint32_t tourLengthElems = alignedAnts;
    const uint32_t bestTourElems   = alignedCities;

    cudaCheck( hipMallocManaged(&randState,      randStateElems  * sizeof(hiprandStateXORWOW_t)) );
    cudaCheck( hipMallocManaged(&edges,          edgesElems      * sizeof(float))               );
    cudaCheck( hipMallocManaged(&eta,            etaElems        * sizeof(float))               );
    cudaCheck( hipMallocManaged(&pheromone,      pheromoneElems  * sizeof(float))               );
    cudaCheck( hipMallocManaged(&fitness,        fitnessElems    * sizeof(float))               );
    cudaCheck( hipMallocManaged(&delta,          deltaElems      * sizeof(float))               );
    cudaCheck( hipMallocManaged(&tabu,           tabuElems       * sizeof(uint32_t))            );
    cudaCheck( hipMallocManaged(&tourLength,     tourLengthElems * sizeof(float))               );
    cudaCheck( hipMallocManaged(&bestTour,       bestTourElems   * sizeof(uint32_t))            );
    cudaCheck( hipMallocManaged(&bestTourLength, sizeof(float))                                 );

    const uint32_t totalMemory = (randStateElems  * sizeof(float)    +
                                  edgesElems      * sizeof(float)    +
                                  etaElems        * sizeof(float)    +
                                  pheromoneElems  * sizeof(float)    +
                                  fitnessElems    * sizeof(float)    +
                                  deltaElems      * sizeof(float)    +
                                  tabuElems       * sizeof(uint32_t) +
                                  tourLengthElems * sizeof(float)    +
                                  bestTourElems   * sizeof(uint32_t) +
                                  1               * sizeof(float));

    std::cout << " **** ACO TSP totalMemory **** \tMB " << (totalMemory / 1024.f) / 1024.f<< std::endl;

    *bestTourLength = FLT_MAX;
    const std::vector<D_TYPE> & tspEdges = tsp.getEdges();
    for (uint32_t i = 0; i < nCities; ++i) {
        for (uint32_t j = 0; j < alignedCities; ++j) {
            const uint32_t alignedId = i * alignedCities + j;
            const uint32_t id = i * nCities + j;
            edges[alignedId] = (j < nCities) ? tspEdges[id] : 0.0;
        }
    }

    // Curand 
    const dim3 initRandBlock(32);
    const dim3 initRandGrid( numberOfBlocks(randStateElems, initRandBlock.x) );
    initCurand <<< initRandGrid, initRandBlock >>>(randState, seed, alignedAnts);
    cudaCheck( hipGetLastError() );
    // Eta
    const dim3 initEtaBlock(32);
    const dim3 initEtaGrid( numberOfBlocks(etaElems, initEtaBlock.x) );
    initEta <<<initEtaGrid, initEtaBlock >>>(eta, edges, nCities, alignedCities);
    cudaCheck( hipGetLastError() );
    // Pheromone
    const dim3 initPheroBlock(32);
    const dim3 initPheroGrid( numberOfBlocks(pheromoneElems, initPheroBlock.x) );
    initPheromone <<<initPheroGrid, initPheroBlock>>> (pheromone, valPheromone, nCities, alignedCities, nCities);
    cudaCheck( hipGetLastError() );
    // Delta
    const dim3 initDeltaBlock(32);
    const dim3 initDeltaGrid( numberOfBlocks(deltaElems, initDeltaBlock.x) );
    initDelta <<<initDeltaGrid, initDeltaBlock>>> (delta, nCities, alignedCities);
    cudaCheck( hipGetLastError() );

    startTimer();
    uint32_t epoch = 0;
    do {
        // initDelta        <<<initDeltaGrid, initDeltaBlock         >>>(delta, nCities, alignedCities);
        // Fitness
        const dim3 fitBlock(32);
        const dim3 fitGrid( numberOfBlocks(fitnessElems, fitBlock.x) );
        calcFitness <<<fitGrid, fitBlock >>> (fitness, pheromone, eta, alpha, beta, nCities, alignedCities);
        cudaCheck( hipGetLastError() );

        // Tour
        const dim3 tourGrid( 1);//( nAnts + 3 )/ 4);
        const dim3 tourBlock(32);
        const uint32_t tourShared  = alignedCities  * sizeof(float)    + // p
                                     1              * sizeof(uint32_t) + // k
                                     alignedCities  * sizeof(uint8_t);   // v
        calcTour <<<tourGrid, tourBlock, tourShared>>> (tabu, fitness, nAnts, nCities, alignedCities, randState);
        cudaCheck( hipGetLastError() );

        // TourLength
        const dim3 lenGrid(nAnts);
        const dim3 lenBlock(64);
        const uint32_t lenShared = lenBlock.x / 32 * sizeof(float);
        calcTourLength <<<lenGrid, lenBlock, lenShared>>> (tourLength, edges, tabu, nAnts, alignedCities, nCities);
        cudaCheck( hipGetLastError() );

        // Update best
        const dim3 bestGrid(1);
        const dim3 bestBlock(32);
        updateBestTour <<<bestGrid, bestBlock>>> (bestTour, bestTourLength, tabu, tourLength, nAnts, alignedCities, nCities);
        cudaCheck( hipGetLastError() );
        
        // Update Delta
        const dim3 deltaGrid(nAnts);
        const dim3 deltaBlock(32);
        const uint32_t deltaShared = alignedCities * sizeof(uint32_t);
        updateDelta <<<deltaGrid, deltaBlock, deltaShared>>> (delta, tabu, tourLength, nAnts, alignedCities, nCities, q);
        cudaCheck( hipGetLastError() );

        // Update Pheromone
        const dim3 pheroBlock(32);
        const dim3 pheroGrid( numberOfBlocks(pheromoneElems, pheroBlock.x) );
        updatePheromone <<<pheroGrid, pheroBlock>>> (pheromone, delta, nCities, alignedCities, rho);
        cudaCheck( hipGetLastError() );
    } while (++epoch < maxEpoch);

    hipDeviceSynchronize();

    printMatrix("tabu", tabu, nAnts, alignedCities);

    stopAndPrintTimer();
    printMatrix("bestTour", bestTour, 1, nCities);
    printResult(tsp.getName(), 0, 0, maxEpoch, getTimerMS(), getTimerUS(), *bestTourLength, !tsp.checkPath(bestTour));

    hipFree(randState);
    hipFree(edges);
    hipFree(eta);
    hipFree(pheromone);
    hipFree(fitness);
    hipFree(delta);
    hipFree(tabu);
    hipFree(tourLength);
    hipFree(bestTour);
    hipFree(bestTourLength);

    return 0;
}
